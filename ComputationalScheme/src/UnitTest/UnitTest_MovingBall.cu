#include "hip/hip_runtime.h"
#include "UnitTest_MovingBall.hpp"

typedef UnitTest_MovingBall::Cell Cell;

__global__ void performGPUSimulationStep_kernel(Cell* cu_field, Cell* cu_lr_halo,
		Cell* cu_tb_halo, Cell* cu_lrtb_halo, size_t N_X, size_t N_Y)
{
	size_t tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    STRUCT_DATA_TYPE data = 1000;
	Cell* C = nullptr;
    Cell* C1 = nullptr;
    C = &cu_field[tid_y * N_X + tid_x];
    if(tid_x == N_X-1) {
        data = cu_lr_halo[N_Y + tid_y].r;
    } else {
        C1 = &cu_field[tid_y * N_X + tid_x + 1];
        data = C1->r;
    }
    __syncthreads();
    C->r = data;
}

__global__ void updateGPUGlobalBorders_kernel(Cell* cu_field, Cell* cu_lr_halo,
		Cell* cu_tb_halo, Cell* cu_lrtb_halo, size_t N_X, size_t N_Y, size_t type)
{
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(type == CU_LEFT_BORDER) {
		cu_lr_halo[tid].r = cu_field[tid * N_X].r;
	} else if(type == CU_RIGHT_BORDER) {
        cu_lr_halo[N_Y + tid].r = 0.0;
	}
}

void* UnitTest_MovingBall::createField(size_t N_X, size_t N_Y)
{
    return (void*)(new Cell[N_X * N_Y]);
}

void* UnitTest_MovingBall::createPageLockedField(size_t N_X, size_t N_Y)
{
    Cell* ptr;
    HANDLE_CUERROR_PTR( hipHostAlloc((void**)&ptr, N_X * N_Y * sizeof(Cell), hipHostMallocDefault) );
    return (void*)ptr;
}

void* UnitTest_MovingBall::createGPUField(size_t N_X, size_t N_Y)
{
    Cell* ptr;
    HANDLE_CUERROR_PTR( hipMalloc((void**)&ptr, N_X * N_Y * sizeof(Cell)) );
    return (void*)ptr;
}

ErrorStatus UnitTest_MovingBall::initField(void* field, size_t N_X, size_t N_Y)
{ // Initialization is on the CPU side
    Cell* cfield = (Cell*)field;
	Cell* C = nullptr;
    size_t global;
	float dx, dy;
    for(size_t x = 0; x < N_X; ++x) {
		dx = x / (float)N_X;
        for(size_t y = 0; y < N_Y; ++y) {
            global = y * N_X + x;
			C = &cfield[global];
			dy = y / (float)N_Y;
            /*if( (dy-0.5)*(dy-0.5) + (dx-0.75)*(dx-0.75) < 0.04 ) {
                C->r = 1.0;
            } else {
                C->r = 0.0;
            }*/
            if(x == 0 && y == 0)
                C->r = 0.0;
            else
                C->r = cos(dx) * cos(dy);
		}
    }
	return GPU_SUCCESS;
}

void* UnitTest_MovingBall::initHalos(size_t N)
{
    return (void*)(new Cell[N]);
}

void* UnitTest_MovingBall::initPageLockedHalos(size_t N)
{
    Cell* ptr;
    HANDLE_CUERROR_PTR( hipHostAlloc((void**)&ptr, N * sizeof(Cell), hipHostMallocDefault) );
    return (void*)ptr;
}

void* UnitTest_MovingBall::initGPUHalos(size_t N)
{
    Cell* ptr;
    HANDLE_CUERROR_PTR( hipMalloc((void**)&ptr, N * sizeof(Cell)) );
    return (void*)ptr;
}

ErrorStatus UnitTest_MovingBall::performCPUSimulationStep(void* tmpCPUField, void* lr_halo,
        void* tb_halo, void* lrtb_halo, size_t N_X, size_t N_Y)
{
	return GPU_SUCCESS;
}

ErrorStatus UnitTest_MovingBall::performGPUSimulationStep(void* cu_field, void* cu_lr_halo,
        void* cu_tb_halo, void* cu_lrtb_halo, size_t N_X, size_t N_Y,
        size_t CUDA_X_BLOCKS, size_t CUDA_Y_BLOCKS, size_t CUDA_X_THREADS,
        size_t CUDA_Y_THREADS, void* stream)
{
	size_t SharedMemoryPerBlock = 0;
    hipStream_t* cuStream = (hipStream_t*)stream;
	/// Launch the CUDA kernel
	performGPUSimulationStep_kernel <<< dim3(CUDA_X_BLOCKS, CUDA_Y_BLOCKS, 1),
		dim3(CUDA_X_THREADS, CUDA_Y_THREADS, 1), SharedMemoryPerBlock,
		*cuStream >>> ((Cell*)cu_field, (Cell*)cu_lr_halo, (Cell*)cu_tb_halo,
            (Cell*)cu_lrtb_halo, N_X, N_Y);
	/// Check if the kernel executed without errors
	lastCudaError = hipGetLastError();
	if(lastCudaError != hipSuccess) {
		errorString = std::string("performGPUSimulationStep: ") +
			std::string(hipGetErrorString(lastCudaError));
		return GPU_ERROR;
	}
	return GPU_SUCCESS;
}

ErrorStatus UnitTest_MovingBall::updateGPUGlobalBorders(void* cu_field, void* cu_lr_halo,
            void* cu_tb_halo, void* cu_lrtb_halo, size_t N_X, size_t N_Y,
            size_t type, size_t CUDA_X_BLOCKS, size_t CUDA_Y_BLOCKS,
            size_t CUDA_X_THREADS, size_t CUDA_Y_THREADS, void* stream)
{
	/// Calculate the amount of shared memory that is required for the kernel
	size_t sharedMemory = 0;
	hipStream_t* cuStream = (hipStream_t*)stream;
	/// Launch the CUDA kernel
	updateGPUGlobalBorders_kernel <<< dim3(CUDA_X_BLOCKS, CUDA_Y_BLOCKS, 1),
		dim3(CUDA_X_THREADS, CUDA_Y_THREADS, 1), sharedMemory,
		*cuStream >>> ((Cell*)cu_field, (Cell*)cu_lr_halo, (Cell*)cu_tb_halo,
            (Cell*)cu_lrtb_halo, N_X, N_Y, type);
	/// Check if the kernel executed without errors
	lastCudaError = hipGetLastError();
	if(lastCudaError != hipSuccess) {
		errorString = std::string("updateGPUGlobalBorders: ") +
			std::string(hipGetErrorString(lastCudaError));
		return GPU_ERROR;
	}
	return GPU_SUCCESS;
}

void* UnitTest_MovingBall::getMarkerValue()
{
    return (void*)(&marker);
}
