#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

/*
 * File:   LatticeBoltzmannModel.cpp
 * Author: eugene
 *
 * Created on November 1, 2017, 2:20 PM
 */

#include "GPUComputationalModel.hpp"
#include <string>

typedef char byte;

GPUComputationalModel::GPUComputationalModel(const char* compModel, const char* gridModel):
    ComputationalModel(compModel, gridModel)
{
    cu_field = nullptr;
    cu_lr_halo = nullptr;
    cu_tb_halo = nullptr;
    cu_lrtb_halo = nullptr;
    snd_cu_lr_halo = nullptr;
    snd_cu_tb_halo = nullptr;
    snd_cu_lrtb_halo = nullptr;
}

GPUComputationalModel::~GPUComputationalModel()
{
}

ErrorStatus GPUComputationalModel::initializeEnvironment()
{
    if(nodeType == NODE_TYPE::COMPUTATIONAL_NODE) {
        /// Initialize the scheme
        CM_HANDLE_GPUERROR(scheme->initScheme());
        /// Allocate page-locked memory for asynchronous data transferring
        /// between GPU and CPU.
        tmpCPUField = scheme->createPageLockedField(lN_X, lN_Y);
        #ifdef __DEBUG__
            dbg_field = scheme->createPageLockedField(lN_X, lN_Y);
        #endif
        CM_HANDLE_GPUERROR_PTR(tmpCPUField);
        lr_halo = scheme->initPageLockedHalos(2*lN_Y);
        CM_HANDLE_GPUERROR_PTR(lr_halo);
        tb_halo = scheme->initPageLockedHalos(2*lN_X);
        CM_HANDLE_GPUERROR_PTR(tb_halo);
        lrtb_halo = scheme->initPageLockedHalos(4);
        CM_HANDLE_GPUERROR_PTR(lrtb_halo);
        rcv_lr_halo = scheme->initPageLockedHalos(2*lN_Y);
        CM_HANDLE_GPUERROR_PTR(rcv_lr_halo);
        rcv_tb_halo = scheme->initPageLockedHalos(2*lN_X);
        CM_HANDLE_GPUERROR_PTR(rcv_tb_halo);
        rcv_lrtb_halo = scheme->initPageLockedHalos(4);
        CM_HANDLE_GPUERROR_PTR(rcv_lrtb_halo);
        /// Initialize CUDA streams
        HANDLE_CUERROR(hipStreamCreate(&streamInternal));
        HANDLE_CUERROR(hipStreamCreate(&streamHaloBorder));
        /// Allocate memory for GPU variables
        cu_field = scheme->createGPUField(lN_X, lN_Y);
        CM_HANDLE_GPUERROR_PTR(cu_field);
        cu_lr_halo = scheme->initGPUHalos(2*lN_Y);
        CM_HANDLE_GPUERROR_PTR(cu_lr_halo);
        cu_tb_halo = scheme->initGPUHalos(2*lN_X);
        CM_HANDLE_GPUERROR_PTR(cu_tb_halo);
        cu_lrtb_halo = scheme->initGPUHalos(4);
        CM_HANDLE_GPUERROR_PTR(cu_lrtb_halo);

        snd_cu_lr_halo = scheme->initGPUHalos(2*lN_Y);
        CM_HANDLE_GPUERROR_PTR(snd_cu_lr_halo);
        snd_cu_tb_halo = scheme->initGPUHalos(2*lN_X);
        CM_HANDLE_GPUERROR_PTR(snd_cu_tb_halo);
        snd_cu_lrtb_halo = scheme->initGPUHalos(4);
        CM_HANDLE_GPUERROR_PTR(snd_cu_lrtb_halo);
        #ifdef __DEBUG__
            dbg_lr_halo = scheme->initPageLockedHalos(2*lN_Y);
            dbg_tb_halo = scheme->initPageLockedHalos(2*lN_X);
            dbg_lrtb_halo = scheme->initPageLockedHalos(4);
        #endif
    } else { // NODE_TYPE::SERVER_NODE
        field = scheme->createField(N_X, N_Y);
        tmpCPUField = scheme->createField(lN_X, lN_Y);
    }
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::updateGlobalField(size_t mpi_node_x, size_t mpi_node_y)
{
	if(nodeType != NODE_TYPE::SERVER_NODE) {
		errorString = "GPUComputationalModel::updateGlobalField: "
			"This function should not be called by a Computational Node";
        return GPU_ERROR;
    }
	memcpyField(mpi_node_x, mpi_node_y, TmpCPUFieldToField);
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::prepareSubfield(size_t mpi_node_x, size_t mpi_node_y)
{
	if(nodeType == NODE_TYPE::COMPUTATIONAL_NODE) {
		size_t sizeOfStruct = scheme->getSizeOfDatastruct();
		size_t field_size = lN_X * lN_Y;
		HANDLE_CUERROR(hipMemcpyAsync(tmpCPUField, cu_field, field_size * sizeOfStruct, hipMemcpyDeviceToHost, streamInternal));
		*Log << std::string("Request for the stream 'streamInternal' to transfer array of ") +
			std::to_string(field_size) + std::string(" field elements from device to host has been placed.");
	} else {
		memcpyField(mpi_node_x, mpi_node_y, FieldToTmpCPUField);
	}
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::loadSubFieldToGPU()
{
	size_t sizeOfStruct = scheme->getSizeOfDatastruct();
	size_t field_size = lN_X * lN_Y;
	HANDLE_CUERROR(hipMemcpyAsync(cu_field, tmpCPUField, field_size * sizeOfStruct, hipMemcpyHostToDevice, streamInternal));
	*Log << std::string("Request for the stream 'streamInternal' to transfer array of ") +
		std::to_string(field_size) + std::string(" field elements from host to device has been placed.");
    #ifdef __DEBUG__
        cpu_memcpy(dbg_field, tmpCPUField, field_size * sizeOfStruct);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_field, cu_field, 0, field_size * sizeOfStruct, *Log);
    #endif
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::gpuSync()
{
	HANDLE_CUERROR(hipStreamSynchronize(streamInternal));
	HANDLE_CUERROR(hipStreamSynchronize(streamHaloBorder));
	HANDLE_CUERROR(hipDeviceSynchronize());
	*Log << "CUDA streams (GPU device) have been successfully synchronized";
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::performSimulationStep()
{
    if(nodeType != NODE_TYPE::COMPUTATIONAL_NODE) {
		errorString = "GPUComputationalModel::performSimulationStep: "
			"This function should not be called by the Server Node";
        return GPU_ERROR;
    }
    #ifdef __DEBUG__
        {
            size_t numberOfBytes = 2*lN_Y * scheme->getSizeOfDatastruct();
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, cu_lr_halo, 0, numberOfBytes, *Log);
        }
    #endif
	CM_HANDLE_GPUERROR(scheme->performGPUSimulationStep(cu_field, cu_lr_halo,
        cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CUDA_X_THREADS, CUDA_Y_THREADS,
		&streamInternal));
    #ifdef __DEBUG__
        size_t numberOfBytes = lN_X * lN_Y * scheme->getSizeOfDatastruct();
        scheme->dbg_performSimulationStep(dbg_field, dbg_lr_halo,
            dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CUDA_X_THREADS, CUDA_Y_THREADS,
    		&streamInternal);
        /**********************************************************************/
        //HANDLE_CUERROR(hipMemcpyAsync(cu_field, dbg_field, lN_X*lN_Y * scheme->getSizeOfDatastruct(), hipMemcpyHostToDevice, streamInternal));
        //gpuSync();
        /**********************************************************************/
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_field, cu_field, 0, numberOfBytes, *Log);
        //_DEBUG_PRINT_FIELDS_CPU_GPU(dbg_field, cu_field, lN_X, lN_Y, scheme->getSizeOfDatastruct(), *Log);
    #endif
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::updateHaloBorderElements(size_t mpi_node_x, size_t mpi_node_y)
{
    if(nodeType != NODE_TYPE::COMPUTATIONAL_NODE) {
		errorString = "GPUComputationalModel::updateHaloBorderElements: "
			"This function should not be called by the Server Node";
        return GPU_ERROR;
    }
    #ifdef __DEBUG__
        //_DEBUG_PRINT_FIELDS_CPU_GPU(dbg_field, cu_field, lN_X, lN_Y, scheme->getSizeOfDatastruct(), *Log);
    #endif
	size_t sizeOfStruct = scheme->getSizeOfDatastruct();
	/// Upload left-right halo elements
	size_t lr_size = 2 * lN_Y;
	HANDLE_CUERROR(hipMemcpyAsync(cu_lr_halo, rcv_lr_halo, lr_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
        std::to_string(lr_size) + std::string(" lr_halo elements from host to device has been placed.");
    #ifdef __DEBUG__
        cpu_memcpy(dbg_lr_halo, rcv_lr_halo, lr_size);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, cu_lr_halo, 0, lr_size * sizeOfStruct, *Log);
    #endif
	/// Upload top-bottom halo elements
	size_t tb_size = 2 * lN_X;
	HANDLE_CUERROR(hipMemcpyAsync(cu_tb_halo, rcv_tb_halo, tb_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
		std::to_string(tb_size) + std::string(" tb_halo elements from host to device has been placed.");
    #ifdef __DEBUG__
        cpu_memcpy(dbg_tb_halo, rcv_tb_halo, tb_size);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_tb_halo, cu_tb_halo, 0, tb_size * sizeOfStruct, *Log);
    #endif
	/// Upload diagonal halo elements
	size_t lrtb_size = 4;
	HANDLE_CUERROR(hipMemcpyAsync(cu_lrtb_halo, rcv_lrtb_halo, lrtb_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
		std::to_string(lrtb_size) + std::string(" lrtb_halo (diagonal) elements from host to device has been placed.");
    #ifdef __DEBUG__
        cpu_memcpy(dbg_lrtb_halo, rcv_lrtb_halo, lrtb_size);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo, 0, lrtb_size * sizeOfStruct, *Log);
    #endif
	/// Update global borders
	if(mpi_node_x == 0) {
		/// Update global left border
		CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_LEFT_BORDER,
			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            size_t numberOfBytes = lN_Y * sizeOfStruct;
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_LEFT_BORDER,
    			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, cu_lr_halo, 0, numberOfBytes, *Log);
        #endif
	}
    if(mpi_node_x == (MPI_NODES_X - 1)) {
		/// Update global right border
		CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BORDER,
			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            size_t numberOfBytes = lN_Y * sizeOfStruct;
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BORDER,
    			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, cu_lr_halo,
                lN_Y * sizeOfStruct, numberOfBytes, *Log);
        #endif
	}
	if(mpi_node_y == 0) {
		/// Update global top border
		CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_TOP_BORDER,
			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            size_t numberOfBytes = lN_X * sizeOfStruct;
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_TOP_BORDER,
    			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_tb_halo, cu_tb_halo, 0,
                numberOfBytes, *Log);
        #endif
	}
    if(mpi_node_y == (MPI_NODES_Y - 1)) {
		/// Update global bottom border
		CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_BOTTOM_BORDER,
			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            size_t numberOfBytes = lN_X * sizeOfStruct;
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_BOTTOM_BORDER,
    			CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_tb_halo, cu_tb_halo,
                lN_X * sizeOfStruct, numberOfBytes, *Log);
        #endif
	}
    /// Update global diagonal border elements
    if(mpi_node_y == 0) {
        /// First row of nodes have the top global border which means that
        /// they have to update top diagonal border elements
        /// Update the left-top diagonal element
    	CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_LEFT_TOP_BORDER,
    		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_LEFT_TOP_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                (CU_LEFT_TOP_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
        #endif
        /// Update the right-top diagonal element
        CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_RIGHT_TOP_BORDER,
    		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_RIGHT_TOP_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                (CU_RIGHT_TOP_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
        #endif
    }
    if(mpi_node_y == (MPI_NODES_Y - 1)) {
        /// Last row of nodes have the bottom global border which means that
        /// they have to update bottom diagonal border elements
        /// Update the left-bottom diagonal element
    	CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_LEFT_BOTTOM_BORDER,
    		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_LEFT_BOTTOM_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                (CU_LEFT_BOTTOM_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
        #endif
        /// Update the right-bottom diagonal element
        CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
            cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BOTTOM_BORDER,
    		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
        #ifdef __DEBUG__
            scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BOTTOM_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
            CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                (CU_RIGHT_BOTTOM_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
        #endif
    }
    if(mpi_node_x == 0) {
        /// First column of nodes have the left global border which means that
        /// they have to update left-top and left-bottom diagonal border elements
        /// Update the left-top diagonal element
        if(mpi_node_y != 0) {
            /// Since the first block has already updated the left-top diagonal element
            CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
                cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_LEFT_TOP_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
            #ifdef __DEBUG__
                scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                    dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_LEFT_TOP_BORDER,
            		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
                CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                    (CU_LEFT_TOP_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
            #endif
        }
        if(mpi_node_y != (MPI_NODES_Y - 1)) {
            /// Since the last block has already updated the left-bottom diagonal element
            CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
                cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_LEFT_BOTTOM_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
            #ifdef __DEBUG__
                scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                    dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_LEFT_BOTTOM_BORDER,
            		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
                CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                    (CU_LEFT_BOTTOM_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
            #endif
        }
    }
    if(mpi_node_x == (MPI_NODES_X - 1)) {
        /// Last column of nodes have the right global border which means that
        /// they have to update right-top and right-bottom diagonal border elements
        /// Update the right-top diagonal element
        if(mpi_node_y != 0) {
            /// Since the first block has already updated the right-top diagonal element
            CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
                cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_RIGHT_TOP_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
            #ifdef __DEBUG__
                scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                    dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_RIGHT_TOP_BORDER,
            		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
                CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                    (CU_RIGHT_TOP_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
            #endif
        }
        /// Update the right-bottom diagonal element
        if(mpi_node_y != (MPI_NODES_Y - 1)) {
            /// Since the last block has already updated the right-bottom diagonal element
            CM_HANDLE_GPUERROR(scheme->updateGPUGlobalBorders(cu_field, cu_lr_halo,
                cu_tb_halo, cu_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BOTTOM_BORDER,
        		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder));
            #ifdef __DEBUG__
                scheme->dbg_updateGlobalBorders(dbg_field, dbg_lr_halo,
                    dbg_tb_halo, dbg_lrtb_halo, lN_X, lN_Y, CU_RIGHT_BOTTOM_BORDER,
            		CUDA_X_THREADS, CUDA_Y_THREADS, &streamHaloBorder);
                CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo,
                    (CU_RIGHT_BOTTOM_BORDER - CU_LEFT_TOP_BORDER) * sizeOfStruct, sizeOfStruct, *Log);
            #endif
        }
    }
    #ifdef __DEBUG__
    {
        size_t sizeOfStruct = scheme->getSizeOfDatastruct();
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, cu_lr_halo, 0, 2*lN_Y*sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_GPU(dbg_lr_halo, cu_lr_halo, 2*lN_Y, sizeOfStruct, *Log);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_tb_halo, cu_tb_halo, 0, 2*lN_X*sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_GPU(dbg_tb_halo, cu_tb_halo, 2*lN_X, sizeOfStruct, *Log);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, cu_lrtb_halo, 0, 4*sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_GPU(dbg_lrtb_halo, cu_lrtb_halo, 4, sizeOfStruct, *Log);
    }
    #endif
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::prepareHaloElements()
{
    #ifdef __DEBUG__
        //_DEBUG_PRINT_FIELDS_CPU_GPU(dbg_field, cu_field, lN_X, lN_Y, scheme->getSizeOfDatastruct(), *Log);
    #endif
	size_t sizeOfStruct = scheme->getSizeOfDatastruct();
	size_t lr_size = 2 * lN_Y;
    size_t tb_size = 2 * lN_X;
    size_t lrtb_size = 4;
    /// Update the halo elements on the GPU
    CM_HANDLE_GPUERROR(updateGPUHaloElements(lr_size, tb_size, lrtb_size));
    /// Send requests to transfer updated halos from the GPU to the CPU
	HANDLE_CUERROR(hipMemcpyAsync(lr_halo, snd_cu_lr_halo, lr_size * sizeOfStruct, hipMemcpyDeviceToHost, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
		std::to_string(lr_size) + std::string(" lr_halo elements from device to host has been placed.");
    #ifdef __DEBUG__
        HANDLE_CUERROR(hipDeviceSynchronize());
        CHECK_CPU_CPU_ARRAYS_EQUALITY_BYTES(lr_halo, dbg_lr_halo, lr_size * sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_CPU(dbg_lr_halo, lr_halo, lr_size, sizeOfStruct, *Log);
    #endif
	HANDLE_CUERROR(hipMemcpyAsync(tb_halo, snd_cu_tb_halo, tb_size * sizeOfStruct, hipMemcpyDeviceToHost, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
		std::to_string(tb_size) + std::string(" tb_halo elements from device to host has been placed.");
    #ifdef __DEBUG__
        HANDLE_CUERROR(hipDeviceSynchronize());
        CHECK_CPU_CPU_ARRAYS_EQUALITY_BYTES(tb_halo, dbg_tb_halo, tb_size * sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_CPU(dbg_tb_halo, tb_halo, tb_size, sizeOfStruct, *Log);
    #endif
	HANDLE_CUERROR(hipMemcpyAsync(lrtb_halo, snd_cu_lrtb_halo, lrtb_size * sizeOfStruct, hipMemcpyDeviceToHost, streamHaloBorder));
	*Log << std::string("Request for the stream 'streamHaloBorder' to transfer array of ") +
		std::to_string(lrtb_size) + std::string(" lrtb_halo (diagonal) elements from device to host has been placed.");
    #ifdef __DEBUG__
        HANDLE_CUERROR(hipDeviceSynchronize());
        CHECK_CPU_CPU_ARRAYS_EQUALITY_BYTES(lrtb_halo, dbg_lrtb_halo, lrtb_size * sizeOfStruct, *Log);
        //_DEBUG_PRINT_ARRAYS_CPU_CPU(dbg_lrtb_halo, lrtb_halo, lrtb_size, sizeOfStruct, *Log);
    #endif
    return GPU_SUCCESS;
}

ErrorStatus GPUComputationalModel::deinitModel()
{
    if(field != nullptr)
        delete[] (byte*)field;
    if(tmpCPUField != nullptr) {
    	if(nodeType == NODE_TYPE::COMPUTATIONAL_NODE) {
    		HANDLE_CUERROR(hipHostFree(tmpCPUField));
    	} else {
    		delete[] (byte*)tmpCPUField;
    	}
    }
    if(lr_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)lr_halo));
    if(tb_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)tb_halo));
    if(lrtb_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)lrtb_halo));
    if(rcv_lr_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)rcv_lr_halo));
    if(rcv_tb_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)rcv_tb_halo));
    if(rcv_lrtb_halo != nullptr)
        HANDLE_CUERROR(hipHostFree((byte*)rcv_lrtb_halo));
    if(cu_field != nullptr)
        HANDLE_CUERROR(hipFree((byte*)cu_field));
    if(cu_lr_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)cu_lr_halo));
    if(cu_tb_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)cu_tb_halo));
    if(cu_lrtb_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)cu_lrtb_halo));
    if(snd_cu_lr_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)snd_cu_lr_halo));
    if(snd_cu_tb_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)snd_cu_tb_halo));
    if(snd_cu_lrtb_halo != nullptr)
        HANDLE_CUERROR(hipFree((byte*)snd_cu_lrtb_halo));
    #ifdef __DEBUG__
        HANDLE_CUERROR(hipHostFree((byte*)dbg_field));
        HANDLE_CUERROR(hipHostFree((byte*)dbg_lr_halo));
        HANDLE_CUERROR(hipHostFree((byte*)dbg_tb_halo));
        HANDLE_CUERROR(hipHostFree((byte*)dbg_lrtb_halo));
    #endif
    return GPU_SUCCESS;
}

__device__ void copyHaloData(byte* to, byte* from, size_t sizeOfStruct)
{
    for(size_t i = 0; i < sizeOfStruct; ++i) {
        to[i] = from[i];
    }
}

/*__global__ void updateGPUHaloElements_kernel(byte* cu_field, byte* snd_cu_lr_halo,
    byte* snd_cu_tb_halo, byte* snd_cu_lrtb_halo, size_t N_X, size_t N_Y, size_t lr_size,
    size_t tb_size, size_t lrtb_size, size_t lr_id, size_t tb_id,
    size_t lrtb_id, size_t totalThreads, size_t sizeOfStruct)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= totalThreads)
        return;
    byte* to = nullptr;
    byte* from = nullptr;
    size_t offset;
    if(tid > lrtb_id) {
        /// Diagonal elements
        // '+ CU_LEFT_TOP_BORDER' is needed because it has a value 4
        tid = tid - lrtb_id + CU_LEFT_TOP_BORDER;
        if(tid == CU_LEFT_TOP_BORDER) {
            to = snd_cu_lrtb_halo + CU_LEFT_TOP_BORDER * sizeOfStruct;
            from = cu_field;
        } else if(tid == CU_RIGHT_TOP_BORDER) {
            to = snd_cu_lrtb_halo + CU_RIGHT_TOP_BORDER * sizeOfStruct;
            from = cu_field + (N_X - 1) * sizeOfStruct;
        } else if(tid == CU_LEFT_BOTTOM_BORDER) {
            to = snd_cu_lrtb_halo + CU_LEFT_BOTTOM_BORDER * sizeOfStruct;
            from = cu_field + N_X * (N_Y - 1) * sizeOfStruct;
        } else { // tid == CU_RIGHT_BOTTOM_BORDER
            to = snd_cu_lrtb_halo + CU_RIGHT_BOTTOM_BORDER * sizeOfStruct;
            from = cu_field + (N_X * N_Y - 1) * sizeOfStruct;
        }
    } else if(tid > tb_id) {
        /// Top-Bottom elements
        tid -= tb_id;
        offset = tid < N_X ? 0 : (N_Y-1) * N_X * sizeOfStruct;
        to = snd_cu_tb_halo + tid * sizeOfStruct;
        from = cu_field + offset + tid * sizeOfStruct;
    } else {
        /// Left-Right elements
        offset = tid < N_Y ? tid * N_X * sizeOfStruct : ((tid + 1) * N_X - 1) * sizeOfStruct;
        to = snd_cu_lr_halo + tid * sizeOfStruct;
        from = cu_field + offset;
    }
    copyHaloData(to, from, sizeOfStruct);
}*/

// TODO: Make a faster version of 'updateGPUHaloElements_kernel'
// Probably, separate computation of each border. Most likely, improve the above version.
__global__ void updateGPUHaloElements_kernel(byte* cu_field, byte* snd_cu_lr_halo,
    byte* snd_cu_tb_halo, byte* snd_cu_lrtb_halo, size_t N_X, size_t N_Y, size_t lr_size,
    size_t tb_size, size_t lrtb_size, size_t totalThreads, size_t sizeOfStruct)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= totalThreads)
        return;
    byte* to = nullptr;
    byte* from = nullptr;
    if(tid < lr_size) {
        to = snd_cu_lr_halo + tid * sizeOfStruct;
        from = cu_field + tid * N_X * sizeOfStruct;
        copyHaloData(to, from, sizeOfStruct);
        to = snd_cu_lr_halo + (N_Y + tid) * sizeOfStruct;
        from = cu_field + ((tid + 1) * N_X - 1) * sizeOfStruct;
        copyHaloData(to, from, sizeOfStruct);
    }
    if(tid < tb_size) {
        to = snd_cu_tb_halo + tid * sizeOfStruct;
        from = cu_field + tid * sizeOfStruct;
        copyHaloData(to, from, sizeOfStruct);
        to = snd_cu_tb_halo + (N_X + tid) * sizeOfStruct;
        from = cu_field + ((N_Y - 1) * N_X + tid) * sizeOfStruct;
        copyHaloData(to, from, sizeOfStruct);
    }
    if(tid < lrtb_size) {
        // '+ CU_LEFT_TOP_BORDER' is needed because it has a value 4
        tid = tid + CU_LEFT_TOP_BORDER;
        if(tid == CU_LEFT_TOP_BORDER) {
            to = snd_cu_lrtb_halo + (tid - CU_LEFT_TOP_BORDER) * sizeOfStruct;
            from = cu_field;
        } else if(tid == CU_RIGHT_TOP_BORDER) {
            to = snd_cu_lrtb_halo + (tid - CU_LEFT_TOP_BORDER) * sizeOfStruct;
            from = cu_field + (N_X - 1) * sizeOfStruct;
        } else if(tid == CU_LEFT_BOTTOM_BORDER) {
            to = snd_cu_lrtb_halo + (tid - CU_LEFT_TOP_BORDER) * sizeOfStruct;
            from = cu_field + N_X * (N_Y - 1) * sizeOfStruct;
        } else { // tid == CU_RIGHT_BOTTOM_BORDER
            to = snd_cu_lrtb_halo + (tid - CU_LEFT_TOP_BORDER) * sizeOfStruct;
            from = cu_field + (N_X * N_Y - 1) * sizeOfStruct;
        }
        copyHaloData(to, from, sizeOfStruct);
    }
}

ErrorStatus GPUComputationalModel::updateGPUHaloElements(size_t lr_size,
    size_t tb_size, size_t lrtb_size)
{
    #ifdef __DEBUG__
        //_DEBUG_PRINT_FIELDS_CPU_GPU(dbg_field, cu_field, lN_X, lN_Y, scheme->getSizeOfDatastruct(), *Log);
    #endif
    size_t TotalThreads = lr_size > tb_size ? (lr_size > lrtb_size ? lr_size : lrtb_size)
                                            : (tb_size > lrtb_size ? tb_size : lrtb_size);
    size_t denom = lr_size > tb_size ? (lr_size > lrtb_size ? CUDA_Y_THREADS : 4)
                                     : (tb_size > lrtb_size ? CUDA_X_THREADS : 4);
    size_t CUDA_X_BLOCKS = (size_t)(floor(((double)TotalThreads-1) / denom + 1));
    //streamHaloBorder
    /// Launch the CUDA kernel
	/*updateGPUHaloElements_kernel <<< dim3(CUDA_X_BLOCKS, 1, 1),
		dim3(CUDA_X_THREADS, 1, 1), 0, streamHaloBorder >>>
            ((byte*)cu_field, (byte*)snd_cu_lr_halo, (byte*)snd_cu_tb_halo,
            (byte*)snd_cu_lrtb_halo, lN_X, lN_Y, lr_size, tb_size, lrtb_size,
            0, lr_size, lr_size + tb_size, TotalThreads, scheme->getSizeOfDatastruct());*/
    updateGPUHaloElements_kernel <<< dim3(CUDA_X_BLOCKS, 1, 1),
		dim3(denom, 1, 1), 0, streamHaloBorder >>>
            ((byte*)cu_field, (byte*)snd_cu_lr_halo, (byte*)snd_cu_tb_halo,
            (byte*)snd_cu_lrtb_halo, lN_X, lN_Y, lr_size, tb_size, lrtb_size,
            TotalThreads, scheme->getSizeOfDatastruct());
    /// Check if the kernel executed without errors
    lastCudaError = hipGetLastError();
    if(lastCudaError != hipSuccess) {
        errorString = std::string("updateGPUGlobalBorders: ") +
            std::string(hipGetErrorString(lastCudaError));
        return GPU_ERROR;
    }
    #ifdef __DEBUG__
        size_t sizeOfStruct = scheme->getSizeOfDatastruct();
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_field, cu_field, 0, lN_X * lN_Y * sizeOfStruct, *Log);
        updateDBGHaloElements(lr_size, tb_size, lrtb_size);
        /**********************************************************************/
        //HANDLE_CUERROR(hipMemcpyAsync(snd_cu_lr_halo, dbg_lr_halo, lr_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
        //HANDLE_CUERROR(hipMemcpyAsync(snd_cu_tb_halo, dbg_tb_halo, tb_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
        //HANDLE_CUERROR(hipMemcpyAsync(snd_cu_lrtb_halo, dbg_lrtb_halo, lrtb_size * sizeOfStruct, hipMemcpyHostToDevice, streamHaloBorder));
        //gpuSync();
        /**********************************************************************/
        //_DEBUG_PRINT_ARRAYS_CPU_GPU(dbg_lr_halo, snd_cu_lr_halo, lr_size, sizeOfStruct, *Log);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lr_halo, snd_cu_lr_halo, 0, lr_size * sizeOfStruct, *Log);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_tb_halo, snd_cu_tb_halo, 0, tb_size * sizeOfStruct, *Log);
        CHECK_CPU_GPU_ARRAYS_EQUALITY_BYTES(dbg_lrtb_halo, snd_cu_lrtb_halo, 0, lrtb_size * sizeOfStruct, *Log);
    #endif
    return GPU_SUCCESS;
}

#ifdef __DEBUG__
ErrorStatus GPUComputationalModel::updateDBGHaloElements(size_t lr_size,
    size_t tb_size, size_t lrtb_size)
{
    size_t sizeOfStruct = scheme->getSizeOfDatastruct();
    byte* dbg_f = (byte*)dbg_field;
    byte* dbg_lr = (byte*)dbg_lr_halo;
    byte* dbg_tb = (byte*)dbg_tb_halo;
    byte* dbg_lrtb = (byte*)dbg_lrtb_halo;
    byte* cell = nullptr;
    byte* lrtb = nullptr;

    for(size_t i = 0; i < lN_Y; ++i) {
        for(size_t s = 0; s < sizeOfStruct; ++s) {
            dbg_lr[i * sizeOfStruct + s] = dbg_f[i * lN_X * sizeOfStruct + s];
            dbg_lr[(lN_Y + i) * sizeOfStruct + s] = dbg_f[((i + 1) * lN_X - 1) * sizeOfStruct + s];
        }
    }

    for(size_t i = 0; i < lN_X; ++i) {
        for(size_t s = 0; s < sizeOfStruct; ++s) {
            dbg_tb[i * sizeOfStruct + s] = dbg_f[i * sizeOfStruct + s];
            dbg_tb[(lN_X + i) * sizeOfStruct + s] = dbg_f[((lN_Y - 1) * lN_X + i) * sizeOfStruct + s];
        }
    }

    for(size_t i = 0; i < lrtb_size; ++i) {
        if(i == 0) { // CU_LEFT_TOP_BORDER
            cell = dbg_f;
            lrtb = dbg_lrtb;
        } else if(i == 1) { // CU_RIGHT_TOP_BORDER
            cell = &dbg_f[(lN_X - 1) * sizeOfStruct];
            lrtb = dbg_lrtb + sizeOfStruct;
        } else if(i == 2) { // CU_LEFT_BOTTOM_BORDER
            cell = &dbg_f[(lN_Y - 1) * lN_X * sizeOfStruct];
            lrtb = dbg_lrtb + 2 * sizeOfStruct;
        } else { // i == 3  // CU_RIGHT_BOTTOM_BORDER
            cell = &dbg_f[(lN_Y * lN_X - 1) * sizeOfStruct];
            lrtb = dbg_lrtb + 3 * sizeOfStruct;
        }
        for(size_t s = 0; s < sizeOfStruct; ++s) {
            lrtb[s] = cell[s];
        }
    }
    return GPU_SUCCESS;
}
#endif
